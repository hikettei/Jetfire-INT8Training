#include "hip/hip_runtime.h"
// #include "include/igemm.h"
#include "../include/common.h"
#include <mma.h>

using namespace nvcuda;

#define OFFSET(row, col, ld) ((row) * (ld) + (col))
#define INT4(pointer) (reinterpret_cast<int4*>(&(pointer))[0])
#define FLOAT2(pointer) (reinterpret_cast<float2*>(&(pointer))[0])
#define FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

// Vanilla INT8 WMMA GEMM
template <typename scalar_t1, typename scalar_t2>
__global__ void igemm_basic_int8_gemm_cuda_kernel(
    scalar_t1 * __restrict__ a, scalar_t1 * __restrict__ b, scalar_t2 * __restrict__ c,
    const int M, const int N, const int K) {

    const int BM = 128;
    const int BN = 256;
    const int BK = 32;

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tid = threadIdx.x;
    int wid = tid >> 5;

    const int APAD = 16;
    const int BPAD = 16; // WARNING: this will cause address misalignment error

    __shared__ int8_t s_a[BM][BK + APAD];
    __shared__ int8_t s_b[BK][BN + BPAD];

    wmma::fragment<wmma::matrix_a, 16, 16, 16, int8_t, wmma::row_major> frag_a[2][4];
    wmma::fragment<wmma::matrix_b, 16, 16, 16, int8_t, wmma::row_major> frag_b[2][4];
    wmma::fragment<wmma::accumulator, 16, 16, 16, int32_t> frag_c[4][4];

    #pragma unroll
    for (int i = 0; i < 4; i++) {
        #pragma unroll
        for (int j = 0; j < 4; j++) {
            wmma::fill_fragment(frag_c[i][j], static_cast<int32_t>(0.0)); // Warning haocheng: this int32_t is strange 
        }
    }

    int load_a_smem_m = (tid >> 1);
    int load_a_smem_k = (tid &  1) << 4;
    int load_b_smem_k = (tid >> 4) << 1;
    int load_b_smem_n = (tid & 15) << 4;

    int load_a_gmem_m = by * BM + load_a_smem_m;
    int load_b_gmem_n = bx * BN + load_b_smem_n;

    int load_a_gmem_addr = OFFSET(load_a_gmem_m, load_a_smem_k, K);
    int load_b_gmem_addr = OFFSET(load_b_smem_k, load_b_gmem_n, N);

    int comp_c_frag_m = wid &  1;
    int comp_c_frag_n = wid >> 1;

    for (int bk = 0; bk < K / BK; bk++) {
        INT4(s_a[load_a_smem_m    ][load_a_smem_k]) = INT4(a[load_a_gmem_addr        ]);
        INT4(s_b[load_b_smem_k    ][load_b_smem_n]) = INT4(b[load_b_gmem_addr        ]);
        INT4(s_b[load_b_smem_k + 1][load_b_smem_n]) = INT4(b[load_b_gmem_addr +     N]);

        load_a_gmem_addr += BK;
        load_b_gmem_addr += BK * N;

        __syncthreads();

        wmma::load_matrix_sync(frag_a[0][0], &s_a[comp_c_frag_m * 64     ][ 0], BK + APAD);
        wmma::load_matrix_sync(frag_a[0][1], &s_a[comp_c_frag_m * 64 + 16][ 0], BK + APAD);
        wmma::load_matrix_sync(frag_a[0][2], &s_a[comp_c_frag_m * 64 + 32][ 0], BK + APAD);
        wmma::load_matrix_sync(frag_a[0][3], &s_a[comp_c_frag_m * 64 + 48][ 0], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][0], &s_a[comp_c_frag_m * 64     ][16], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][1], &s_a[comp_c_frag_m * 64 + 16][16], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][2], &s_a[comp_c_frag_m * 64 + 32][16], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][3], &s_a[comp_c_frag_m * 64 + 48][16], BK + APAD);

        wmma::load_matrix_sync(frag_b[0][0], &s_b[ 0][comp_c_frag_n * 64     ], BN + BPAD);
        wmma::load_matrix_sync(frag_b[0][1], &s_b[ 0][comp_c_frag_n * 64 + 16], BN + BPAD);
        wmma::load_matrix_sync(frag_b[0][2], &s_b[ 0][comp_c_frag_n * 64 + 32], BN + BPAD);
        wmma::load_matrix_sync(frag_b[0][3], &s_b[ 0][comp_c_frag_n * 64 + 48], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][0], &s_b[16][comp_c_frag_n * 64     ], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][1], &s_b[16][comp_c_frag_n * 64 + 16], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][2], &s_b[16][comp_c_frag_n * 64 + 32], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][3], &s_b[16][comp_c_frag_n * 64 + 48], BN + BPAD);

        #pragma unroll
        for (int i = 0; i < 4; i++) {
            #pragma unroll
            for (int j = 0; j < 4; j++) {
                wmma::mma_sync(frag_c[i][j], frag_a[0][i], frag_b[0][j], frag_c[i][j]);
                wmma::mma_sync(frag_c[i][j], frag_a[1][i], frag_b[1][j], frag_c[i][j]);
            }
        }

        __syncthreads();
    }

    int32_t* ch = reinterpret_cast<int32_t*>(c);
    int store_c_gmem_m = by * BM + comp_c_frag_m * 64;
    int store_c_gmem_n = bx * BN + comp_c_frag_n * 64;
    int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);
    #pragma unroll
    for (int i = 0; i < 4; i++) {
        #pragma unroll
        for (int j = 0; j < 4; j++) {
            wmma::store_matrix_sync(&ch[store_c_gmem_addr + i * 16 * N + j * 16], frag_c[i][j], N, wmma::mem_row_major);
        }
    }
}

// Vanilla INT8 WMMA GEMM
torch::Tensor igemm_basic_int8_gemm_cuda(
    torch::Tensor X, torch::Tensor W,
    const int M, const int N, const int K) {
    // X shape (M, K), W shape (K, N)
    
    const int BM = 128, BN = 256;
    dim3 blockDim(256);
    int BX = (N + BN - 1) / BN;
    int BY = (M + BM - 1) / BM;
    dim3 gridDim(BX, BY);

    auto option_output = torch::TensorOptions().dtype(torch::kInt32).device(X.device());
    torch::Tensor O = torch::empty({M, N}, option_output);

    // std::cout << X.scalar_type() << std::endl;
    // std::cout << X.dtype() << std::endl;
    // std::cout << O.scalar_type() << std::endl;

    igemm_basic_int8_gemm_cuda_kernel<int8_t, int32_t><<<gridDim, blockDim>>>(
        X.data_ptr<int8_t>(),
        W.data_ptr<int8_t>(),
        O.data_ptr<int32_t>(),
        M,
        N,
        K
    );
    return O;
}
