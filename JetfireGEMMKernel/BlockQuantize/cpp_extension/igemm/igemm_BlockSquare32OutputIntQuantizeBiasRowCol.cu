#include "hip/hip_runtime.h"
// #include "include/igemm.h"
#include "../include/common.h"
#include <mma.h>

using namespace nvcuda;

#define OFFSET(row, col, ld) ((row) * (ld) + (col))
#define INT1(pointer) (reinterpret_cast<int1*>(&(pointer))[0])
#define INT2(pointer) (reinterpret_cast<int2*>(&(pointer))[0])
#define INT4(pointer) (reinterpret_cast<int4*>(&(pointer))[0])
#define FLOAT2(pointer) (reinterpret_cast<float2*>(&(pointer))[0])
#define FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

__device__ int8_t clamprowcol(int value, int8_t minVal, int8_t maxVal) {
    return (value < minVal) ? minVal : (value > maxVal) ? maxVal : static_cast<int8_t>(value);
}

// Quantize + Dequantize Vanilla INT8 GEMM
template <typename scalar_t1, typename scalar_t2>
__global__ void igemm_output_int_quantize_bias_rowcol_cuda_kernel(
    scalar_t1 * __restrict__ a, scalar_t1 * __restrict__ b, 
    float * __restrict__ bias, float * __restrict__ biasmax,
    half *__restrict__ sa, half *__restrict__ sb,
    int8_t * __restrict__ c, half *__restrict__ sc,
    const int M, const int N, const int K) {

    const int BM = 128;
    const int BN = 128;
    const int BK = 32;
    
    const int QM = 32;
    const int QN = 32;
    const int QK = 32;

    const int BSM = BM / QM;
    const int BSK = BK / QK;
    const int BSN = BN / QN;

    const int NUMQM = M / QM;
    const int NUMQK = K / QK;
    const int NUMQN = N / QN;

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tid = threadIdx.x;
    int tid_mod = tid % 32;
    int wid = tid >> 5;

    if (bx >= N / BN || by >= M / BM)
        return;

    const int APAD = 16;
    const int BPAD = 16; // WARNING: this will cause address misalignment error
    const int ACCPAD = 4;

    extern __shared__ half sharedMem[];
    int8_t (*s_a)[BK + APAD] = reinterpret_cast<int8_t (*)[BK + APAD]>(sharedMem);
    int8_t (*s_b)[BK + BPAD] = reinterpret_cast<int8_t (*)[BK + BPAD]>(&s_a[2 * BM]);
    // __shared__ float thread_max[2][64];

    float (*acc_float)[2 * QN + ACCPAD] = reinterpret_cast<float (*)[2 * QN + ACCPAD]>(&sharedMem);
    float (*thread_max)[64] = reinterpret_cast<float (*)[64]>(&acc_float[2 * QM]);

    int s_a_db_offset = BM * (BK + APAD);
    int s_b_db_offset = BN * (BK + BPAD);

    // __shared__ int8_t s_a[BM][BK + APAD];
    // __shared__ int8_t s_b[BK][BN + BPAD];

    half s_qa[BSM];
    half s_qb[BSN]; // based on that BK == QK == 32

    wmma::fragment<wmma::matrix_a, 16, 16, 16, int8_t, wmma::row_major> frag_a[2][4];
    wmma::fragment<wmma::matrix_b, 16, 16, 16, int8_t, wmma::col_major> frag_b[2][4];
    wmma::fragment<wmma::accumulator, 16, 16, 16, float> frag_fpc[4][4];
    wmma::fragment<wmma::accumulator, 16, 16, 16, int32_t> frag_intc[4][4];

    #pragma unroll
    for (int i = 0; i < 4; i++) {
        #pragma unroll
        for (int j = 0; j < 4; j++) {
            wmma::fill_fragment(frag_fpc[i][j], static_cast<float>(0.0)); // Warning haocheng: this int32_t is strange 
            wmma::fill_fragment(frag_intc[i][j], static_cast<int32_t>(0.0)); // Warning haocheng: this int32_t is strange 
        }
    }

    // input tensor address
    int load_a_smem_m = (tid >> 1) << 1;
    int load_a_smem_k = (tid &  1) << 4;
    int load_b_smem_n = (tid >> 1) << 1;
    int load_b_smem_k = (tid &  1) << 4;

    int s_a_base_addr = __cvta_generic_to_shared(s_a[0]);
    int s_b_base_addr = __cvta_generic_to_shared(s_b[0]);
    int load_a_smem_addr_0 = s_a_base_addr + OFFSET(load_a_smem_m, load_a_smem_k, BK + APAD) * sizeof(int8_t);
    int load_a_smem_addr_1 = load_a_smem_addr_0 + (BK + APAD) * sizeof(int8_t);
    int load_b_smem_addr_0 = s_b_base_addr + OFFSET(load_b_smem_n, load_b_smem_k, BK + BPAD) * sizeof(int8_t);
    int load_b_smem_addr_1 = load_b_smem_addr_0 + (BK + BPAD) * sizeof(int8_t);

    int load_a_gmem_m = by * BM + load_a_smem_m;
    int load_b_gmem_n = bx * BN + load_b_smem_n;

    int load_a_gmem_addr = OFFSET(load_a_gmem_m, load_a_smem_k, K);
    int load_b_gmem_addr = OFFSET(load_b_gmem_n, load_b_smem_k, K);

    int comp_c_frag_m = wid &  1;
    int comp_c_frag_n = wid >> 1;

    // scale factor address
    int load_sa_gmem_m = by * BSM;
    int load_sb_gmem_n = bx * BSN;

    half scale_qa0, scale_qa1, scale_qb0, scale_qb1;
    float scale_qa0f, scale_qa1f, scale_qb0f, scale_qb1f;
    float scale_a, scale_b, scale_ab;

    {
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_a_smem_addr_0), "l"(&a[load_a_gmem_addr        ]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_a_smem_addr_1), "l"(&a[load_a_gmem_addr +     K]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_b_smem_addr_0), "l"(&b[load_b_gmem_addr        ]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_b_smem_addr_1), "l"(&b[load_b_gmem_addr +     K]));

        asm ("cp.async.commit_group;\n" ::);
        asm ("cp.async.wait_group 0;\n" ::);

        __syncthreads();
    }

    #pragma unroll 64
    for (int bk = 1; bk < K / BK; bk++) {
        // INT4(s_a[load_a_smem_m    ][load_a_smem_k]) = INT4(a[load_a_gmem_addr        ]);
        // INT4(s_a[load_a_smem_m + 1][load_a_smem_k]) = INT4(a[load_a_gmem_addr     + K]);
        // INT4(s_b[load_b_smem_k    ][load_b_smem_n]) = INT4(b[load_b_gmem_addr        ]);
        // INT4(s_b[load_b_smem_k + 1][load_b_smem_n]) = INT4(b[load_b_gmem_addr     + N]);

        int smem_sel = (bk & 1) ^ 1;
        int smem_sel_next = ((bk - 1) & 1) ^ 1;

        load_a_gmem_addr += BK;
        load_b_gmem_addr += BK;

        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_a_smem_addr_0 + smem_sel_next * s_a_db_offset * (int)sizeof(int8_t)), "l"(&a[load_a_gmem_addr        ]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_a_smem_addr_1 + smem_sel_next * s_a_db_offset * (int)sizeof(int8_t)), "l"(&a[load_a_gmem_addr +     K]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_b_smem_addr_0 + smem_sel_next * s_b_db_offset * (int)sizeof(int8_t)), "l"(&b[load_b_gmem_addr        ]));
        asm ("cp.async.ca.shared.global [%0], [%1], 16;\n" :
            : "r"(load_b_smem_addr_1 + smem_sel_next * s_b_db_offset * (int)sizeof(int8_t)), "l"(&b[load_b_gmem_addr +     K]));

        wmma::load_matrix_sync(frag_a[0][0], &s_a[smem_sel * BM + comp_c_frag_m * 64     ][ 0], BK + APAD);
        wmma::load_matrix_sync(frag_a[0][1], &s_a[smem_sel * BM + comp_c_frag_m * 64 + 16][ 0], BK + APAD);
        wmma::load_matrix_sync(frag_a[0][2], &s_a[smem_sel * BM + comp_c_frag_m * 64 + 32][ 0], BK + APAD);
        wmma::load_matrix_sync(frag_a[0][3], &s_a[smem_sel * BM + comp_c_frag_m * 64 + 48][ 0], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][0], &s_a[smem_sel * BM + comp_c_frag_m * 64     ][16], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][1], &s_a[smem_sel * BM + comp_c_frag_m * 64 + 16][16], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][2], &s_a[smem_sel * BM + comp_c_frag_m * 64 + 32][16], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][3], &s_a[smem_sel * BM + comp_c_frag_m * 64 + 48][16], BK + APAD);

        wmma::load_matrix_sync(frag_b[0][0], &s_b[smem_sel * BN + comp_c_frag_n * 64     ][ 0], BK + BPAD);
        wmma::load_matrix_sync(frag_b[0][1], &s_b[smem_sel * BN + comp_c_frag_n * 64 + 16][ 0], BK + BPAD);
        wmma::load_matrix_sync(frag_b[0][2], &s_b[smem_sel * BN + comp_c_frag_n * 64 + 32][ 0], BK + BPAD);
        wmma::load_matrix_sync(frag_b[0][3], &s_b[smem_sel * BN + comp_c_frag_n * 64 + 48][ 0], BK + BPAD);
        wmma::load_matrix_sync(frag_b[1][0], &s_b[smem_sel * BN + comp_c_frag_n * 64     ][16], BK + BPAD);
        wmma::load_matrix_sync(frag_b[1][1], &s_b[smem_sel * BN + comp_c_frag_n * 64 + 16][16], BK + BPAD);
        wmma::load_matrix_sync(frag_b[1][2], &s_b[smem_sel * BN + comp_c_frag_n * 64 + 32][16], BK + BPAD);
        wmma::load_matrix_sync(frag_b[1][3], &s_b[smem_sel * BN + comp_c_frag_n * 64 + 48][16], BK + BPAD);

        FLOAT2(s_qa[0]) = FLOAT2(sa[load_sa_gmem_m        ]);
        FLOAT2(s_qb[0]) = FLOAT2(sb[load_sb_gmem_n    ]);

        scale_qa0 = s_qa[2 * comp_c_frag_m    ];
        scale_qa1 = s_qa[2 * comp_c_frag_m + 1];

        scale_qb0 = s_qb[2 * comp_c_frag_n    ];
        scale_qb1 = s_qb[2 * comp_c_frag_n + 1];

        scale_qa0f = __half2float(scale_qa0);
        scale_qa1f = __half2float(scale_qa1); 
        scale_qb0f = __half2float(scale_qb0); 
        scale_qb1f = __half2float(scale_qb1);

        load_sa_gmem_m += BSK * NUMQM;
        load_sb_gmem_n += BSK * NUMQN;

        #pragma unroll
        for (int i = 0; i < 4; i++) {
            if (i == 0 || i == 1) {
                scale_a = scale_qa0f;
            } else if (i == 2 || i == 3) {
                scale_a = scale_qa1f;
            } else{
                printf("Error I = %d", i);
            }
            #pragma unroll
            for (int j = 0; j < 4; j++) {
                if (j == 0 || j == 1) {
                    scale_b = scale_qb0f;
                } else if (j == 2 || j == 3) {
                    scale_b = scale_qb1f;
                } else{
                    printf("Error J = %d", j);
                }

                scale_ab = scale_a * scale_b;
                wmma::fill_fragment(frag_intc[i][j], static_cast<int32_t>(0.0));
                wmma::mma_sync(frag_intc[i][j], frag_a[0][i], frag_b[0][j], frag_intc[i][j]);
                wmma::mma_sync(frag_intc[i][j], frag_a[1][i], frag_b[1][j], frag_intc[i][j]);

                #pragma unroll
                for(int k=0; k < frag_intc[i][j].num_elements; k++) {
                    frag_fpc[i][j].x[k] += scale_ab * frag_intc[i][j].x[k];
                }
            }
        }

        asm ("cp.async.commit_group;\n" ::);
        asm ("cp.async.wait_group 0;\n" ::);

        __syncthreads();
    }

    int smem_sel = ((K / BK) & 1) ^ 1;
    wmma::load_matrix_sync(frag_a[0][0], &s_a[smem_sel * BM + comp_c_frag_m * 64     ][ 0], BK + APAD);
    wmma::load_matrix_sync(frag_a[0][1], &s_a[smem_sel * BM + comp_c_frag_m * 64 + 16][ 0], BK + APAD);
    wmma::load_matrix_sync(frag_a[0][2], &s_a[smem_sel * BM + comp_c_frag_m * 64 + 32][ 0], BK + APAD);
    wmma::load_matrix_sync(frag_a[0][3], &s_a[smem_sel * BM + comp_c_frag_m * 64 + 48][ 0], BK + APAD);
    wmma::load_matrix_sync(frag_a[1][0], &s_a[smem_sel * BM + comp_c_frag_m * 64     ][16], BK + APAD);
    wmma::load_matrix_sync(frag_a[1][1], &s_a[smem_sel * BM + comp_c_frag_m * 64 + 16][16], BK + APAD);
    wmma::load_matrix_sync(frag_a[1][2], &s_a[smem_sel * BM + comp_c_frag_m * 64 + 32][16], BK + APAD);
    wmma::load_matrix_sync(frag_a[1][3], &s_a[smem_sel * BM + comp_c_frag_m * 64 + 48][16], BK + APAD);

    wmma::load_matrix_sync(frag_b[0][0], &s_b[smem_sel * BN + comp_c_frag_n * 64     ][ 0], BK + BPAD);
    wmma::load_matrix_sync(frag_b[0][1], &s_b[smem_sel * BN + comp_c_frag_n * 64 + 16][ 0], BK + BPAD);
    wmma::load_matrix_sync(frag_b[0][2], &s_b[smem_sel * BN + comp_c_frag_n * 64 + 32][ 0], BK + BPAD);
    wmma::load_matrix_sync(frag_b[0][3], &s_b[smem_sel * BN + comp_c_frag_n * 64 + 48][ 0], BK + BPAD);
    wmma::load_matrix_sync(frag_b[1][0], &s_b[smem_sel * BN + comp_c_frag_n * 64     ][16], BK + BPAD);
    wmma::load_matrix_sync(frag_b[1][1], &s_b[smem_sel * BN + comp_c_frag_n * 64 + 16][16], BK + BPAD);
    wmma::load_matrix_sync(frag_b[1][2], &s_b[smem_sel * BN + comp_c_frag_n * 64 + 32][16], BK + BPAD);
    wmma::load_matrix_sync(frag_b[1][3], &s_b[smem_sel * BN + comp_c_frag_n * 64 + 48][16], BK + BPAD);

    FLOAT2(s_qa[0]) = FLOAT2(sa[load_sa_gmem_m        ]);
    FLOAT2(s_qb[0]) = FLOAT2(sb[load_sb_gmem_n    ]);

    scale_qa0 = s_qa[2 * comp_c_frag_m    ];
    scale_qa1 = s_qa[2 * comp_c_frag_m + 1];

    scale_qb0 = s_qb[2 * comp_c_frag_n    ];
    scale_qb1 = s_qb[2 * comp_c_frag_n + 1];

    scale_qa0f = __half2float(scale_qa0);
    scale_qa1f = __half2float(scale_qa1); 
    scale_qb0f = __half2float(scale_qb0); 
    scale_qb1f = __half2float(scale_qb1);

    load_sa_gmem_m += BSK * NUMQM;
    load_sb_gmem_n += BSK * NUMQN;

    #pragma unroll
    for (int i = 0; i < 4; i++) {
        if (i == 0 || i == 1) {
            scale_a = scale_qa0f;
        } else if (i == 2 || i == 3) {
            scale_a = scale_qa1f;
        } else{
            printf("Error I = %d", i);
        }
        #pragma unroll
        for (int j = 0; j < 4; j++) {
            if (j == 0 || j == 1) {
                scale_b = scale_qb0f;
            } else if (j == 2 || j == 3) {
                scale_b = scale_qb1f;
            } else{
                printf("Error J = %d", j);
            }

            scale_ab = scale_a * scale_b;
            wmma::fill_fragment(frag_intc[i][j], static_cast<int32_t>(0.0));
            wmma::mma_sync(frag_intc[i][j], frag_a[0][i], frag_b[0][j], frag_intc[i][j]);
            wmma::mma_sync(frag_intc[i][j], frag_a[1][i], frag_b[1][j], frag_intc[i][j]);

            #pragma unroll
            for(int k=0; k < frag_intc[i][j].num_elements; k++) {
                frag_fpc[i][j].x[k] += scale_ab * frag_intc[i][j].x[k];
            }
        }
    }
    
    __syncthreads();

    // int32_t* ch = reinterpret_cast<int32_t*>(c);
    int store_c_gmem_m = by * BM + comp_c_frag_m * 64;
    int store_c_gmem_n = bx * BN + comp_c_frag_n * 64;
    int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);

    // int32_t* ch = reinterpret_cast<int32_t*>(c);
    int store_sc_gmem_m = by * BSM + comp_c_frag_m * 2;
    int store_sc_gmem_n = bx * BSN + comp_c_frag_n * 2;
    int store_sc_gmem_addr = OFFSET(store_sc_gmem_m, store_sc_gmem_n, NUMQN);
    // printf("tid: %d | by: %d, bx: %d | store_sc_gmem_addr: %d | store_c_gmem_m: %d | store_c_gmem_n: %d | NUMQN: %d \n", 
    //         tid, by, bx, store_sc_gmem_addr, store_c_gmem_m, store_c_gmem_n, NUMQN);


    // output tensor address
    int store_c_smem_m = (tid_mod >> 2) << 2;
    int store_c_smem_n = (tid_mod &  3) << 3;

    float BiasMax = biasmax[0];
    #pragma unroll
    for (int x = 0; x < 4; x += 2) {
        for (int y = 0; y < 4; y += 2) {
            for (int i = 0; i < 2; i++) {
                for (int j = 0; j < 2; j++) {
                    wmma::store_matrix_sync(&acc_float[comp_c_frag_m * 32 + i * 16][comp_c_frag_n * 32 + j * 16], frag_fpc[x + i][y + j], 2 * QN + ACCPAD, wmma::mem_row_major);
                }
            }

            float absMaxVal = -1e10;
            float thd_max = 0;
            for (int k = 0; k < QN; k++) {
                thd_max = fmaxf(thd_max, fabs(acc_float[comp_c_frag_m * 32 + k][comp_c_frag_n * 32 + tid_mod]));
                // if ((tid >= 32) && (tid < 64) && (x == 2) && (y == 0)) printf("tid: %d, k: %d, xx: %d, yy: %d | thd_max = %f \n", tid, k, comp_c_frag_m * 32 + k, comp_c_frag_n * 32 + tid_mod, thd_max);
            }

            // if (tid == 32) printf("tid: %d, xx: %d, yy: %d, thd_max: %f \n", tid, comp_c_frag_m, comp_c_frag_n * 32 + tid_mod, thd_max);
            thread_max[comp_c_frag_m][comp_c_frag_n * 32 + tid_mod] = thd_max;
            __syncwarp();

            for (int active_thread_num = 16; active_thread_num >= 1; active_thread_num /= 2) { 
                if (tid_mod < active_thread_num) { 
                    thread_max[comp_c_frag_m][comp_c_frag_n * 32 + tid_mod] = fmaxf(thread_max[comp_c_frag_m][comp_c_frag_n * 32 + tid_mod], thread_max[comp_c_frag_m][comp_c_frag_n * 32 + tid_mod + active_thread_num]); 
                }
                __syncwarp(); 
            } 

            absMaxVal = thread_max[comp_c_frag_m][comp_c_frag_n * 32] + BiasMax;

            for (int i = 0; i < 4; i++) {
                int8_t ci_val[8];
                float cf_val[8];
                float bias_reg[8];
                
                FLOAT4(cf_val[0]) = FLOAT4(acc_float[comp_c_frag_m * 32 + store_c_smem_m + i][comp_c_frag_n * 32 + store_c_smem_n + 0]);
                FLOAT4(cf_val[4]) = FLOAT4(acc_float[comp_c_frag_m * 32 + store_c_smem_m + i][comp_c_frag_n * 32 + store_c_smem_n + 4]);
                // FLOAT4(cf_val[4]) = FLOAT4(acc_float[comp_c_frag_m * 32 + store_c_smem_m + i][comp_c_frag_n * 32 + store_c_smem_n + 4]);

                FLOAT4(bias_reg[0]) = FLOAT4(bias[store_c_gmem_n + y * 16 + store_c_smem_n + 0]);
                FLOAT4(bias_reg[4]) = FLOAT4(bias[store_c_gmem_n + y * 16 + store_c_smem_n + 4]);

                for (int j = 0; j < 8; j++) {
                    // printf("!!!%d %d %d %f %f\n", __float2int_rn(127 * (cf_val[j] + bias_reg[j]) / absMaxVal), clamp(__float2int_rn(127 * (cf_val[j] + bias_reg[j]) / absMaxVal), -127, 127), __float2int_rn(127 * cf_val[j] / absMaxVal), cf_val[j], bias_reg[j]);
                    ci_val[j] = clamprowcol(__float2int_rn(127 * (cf_val[j] + bias_reg[j]) / absMaxVal), -127, 127);
                    // if (tid == 32) printf("tid: %d | c_val: %d | absMaxVal: %f | acc_float: %f | xx: %d | yy: %d | store_xy: %d \n", 
                    //                        tid, c_val, absMaxVal, acc_float[comp_c_frag_m * 32 + store_c_smem_m + i][comp_c_frag_n * 32 + store_c_smem_n + j],
                    //                        comp_c_frag_m * 32 + store_c_smem_m + i, comp_c_frag_n * 32 + store_c_smem_n + j,
                    //                        store_c_gmem_m + (x * 16 + store_c_smem_m + i) * N + (y * 16 + store_c_smem_n + j));
                }
                INT2(c[store_c_gmem_addr + (x * 16 + store_c_smem_m + i) * N + (y * 16 + store_c_smem_n)]) = INT2(ci_val[0]);
            }

            // printf("line: %d, tid: %d, store_sc_gmem_addr: %d, x: %d, y: %d, absMaxVal: %f, Final Address: %d \n", __LINE__, tid, store_sc_gmem_addr, x, y, absMaxVal, store_sc_gmem_addr + (x / 2) * NUMQN + (y / 2));
            sc[store_sc_gmem_addr + (x / 2) * NUMQN + (y / 2)] = __float2half(absMaxVal / 127);
            // printf("%d \n", __LINE__);
        }
    }

    // // int32_t* ch = reinterpret_cast<int32_t*>(c);
    // int store_c_gmem_m = by * BM + comp_c_frag_m * 64;
    // int store_c_gmem_n = bx * BN + comp_c_frag_n * 64;
    // int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);
    // #pragma unroll
    // for (int i = 0; i < 4; i++) {
    //     #pragma unroll
    //     for (int j = 0; j < 4; j++) {
    //         wmma::store_matrix_sync(&c[store_c_gmem_addr + i * 16 * N + j * 16], frag_fpc[i][j], N, wmma::mem_row_major);
    //     }
    // }
}

// Quantize + Dequantize Vanilla INT8 GEMM
std::tuple<torch::Tensor, torch::Tensor> igemm_output_int_quantize_bias_rowcol_cuda(
    torch::Tensor X, torch::Tensor W, torch::Tensor bias, torch::Tensor biasmax,
    torch::Tensor SX, torch::Tensor SW,
    const int M, const int N, const int K) {
    // X shape (M, K), W shape (K, N)
    
    const int BM = 128, BN = 128;
    dim3 blockDim(128);
    int BX = (N + BN - 1) / BN;
    int BY = (M + BM - 1) / BM;
    const int NSPLIT = 4096;
    int split_num = (N + NSPLIT - 1) / NSPLIT;
    dim3 gridDim((BX + split_num - 1) / split_num, BY, split_num);

    int QM = SX.size(1);
    int QK = SX.size(0);
    int QN = SW.size(1);

    auto option_output = torch::TensorOptions().dtype(torch::kInt8).device(X.device());
    torch::Tensor O = torch::empty({M, N}, option_output);

    auto option_scale_output = torch::TensorOptions().dtype(torch::kFloat16).device(X.device());
    torch::Tensor SO = torch::empty({QM, QN}, option_scale_output);

    // std::cout << X.scalar_type() << std::endl;
    // std::cout << X.dtype() << std::endl;
    // std::cout << O.scalar_type() << std::endl;

    // std::cout << SX << std::endl;
    // std::cout << SW << std::endl;

    int maxSmem = 30 * 1024;
    hipFuncSetAttribute(reinterpret_cast<const void*>(igemm_output_int_quantize_bias_rowcol_cuda_kernel<int8_t), int8_t>, hipFuncAttributeMaxDynamicSharedMemorySize, maxSmem);

    igemm_output_int_quantize_bias_rowcol_cuda_kernel<int8_t, int8_t><<<gridDim, blockDim, maxSmem>>>(
        X.data_ptr<int8_t>(),
        W.data_ptr<int8_t>(),
        bias.data_ptr<float>(),
        biasmax.data_ptr<float>(),
        (half*)SX.data_ptr<at::Half>(),
        (half*)SW.data_ptr<at::Half>(),
        O.data_ptr<int8_t>(),
        (half*)SO.data_ptr<at::Half>(),
        M,
        N,
        K
    );
    return std::make_tuple(O, SO);
}