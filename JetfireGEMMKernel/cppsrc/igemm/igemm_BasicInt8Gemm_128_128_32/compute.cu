#include "hip/hip_runtime.h"
// #include "include/igemm.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <vector>
#include <random>
#include <hiprand.h>
#include <float.h>
#include <hiprand/hiprand_kernel.h>
#include <mma.h>

using namespace nvcuda;

#define OFFSET(row, col, ld) ((row) * (ld) + (col))
#define INT4(pointer) (reinterpret_cast<int4*>(&(pointer))[0])
#define FLOAT2(pointer) (reinterpret_cast<float2*>(&(pointer))[0])
#define FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

void cpuI8I32Gemm(int8_t *a, int8_t *b, int32_t *c, int M, int N, int K) {

    for (int m = 0; m < M; m++) {
        for (int n = 0; n < N; n++) {
            float psum = 0.0;
            for (int k = 0; k < K; k++) {
                psum += (float)a[OFFSET(m, k, K)] * (float)b[OFFSET(k, n, N)];
            }
            c[OFFSET(m, n, N)] = (int32_t)psum;
        }
    }
}

// Vanilla INT8 WMMA GEMM
template <typename scalar_t1, typename scalar_t2>
__global__ void igemm_basic_int8_gemm_cuda_kernel(
    scalar_t1 * __restrict__ a, scalar_t1 * __restrict__ b, scalar_t2 * __restrict__ c,
    const int M, const int N, const int K) {

    const int BM = 128;
    const int BN = 128;
    const int BK = 32;

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tid = threadIdx.x;
    int wid = tid >> 5;

    const int APAD = 16;
    const int BPAD = 16; // WARNING: this will cause address misalignment error

    __shared__ int8_t s_a[BM][BK + APAD];
    __shared__ int8_t s_b[BK][BN + BPAD];

    wmma::fragment<wmma::matrix_a, 16, 16, 16, int8_t, wmma::row_major> frag_a[2][4]; // 64 * 32
    wmma::fragment<wmma::matrix_b, 16, 16, 16, int8_t, wmma::row_major> frag_b[2][4]; // 32 * 64
    wmma::fragment<wmma::accumulator, 16, 16, 16, int32_t> frag_c[4][4];

    #pragma unroll
    for (int i = 0; i < 4; i++) {
        #pragma unroll
        for (int j = 0; j < 4; j++) {
            wmma::fill_fragment(frag_c[i][j], static_cast<int32_t>(0.0)); // Warning haocheng: this int32_t is strange 
        }
    }

    // 128 thread, one load 32 element 
    int load_a_smem_m = (tid >> 1) << 1;
    int load_a_smem_k = (tid &  1) << 4;
    int load_b_smem_k = (tid >> 3) << 1;
    int load_b_smem_n = (tid &  7) << 4;

    int load_a_gmem_m = by * BM + load_a_smem_m;
    int load_b_gmem_n = bx * BN + load_b_smem_n;

    int load_a_gmem_addr = OFFSET(load_a_gmem_m, load_a_smem_k, K);
    int load_b_gmem_addr = OFFSET(load_b_smem_k, load_b_gmem_n, N);

    int comp_c_frag_m = wid &  1;
    int comp_c_frag_n = wid >> 1;

    for (int bk = 0; bk < K / BK; bk++) {
        INT4(s_a[load_a_smem_m    ][load_a_smem_k]) = INT4(a[load_a_gmem_addr        ]);
        INT4(s_a[load_a_smem_m + 1][load_a_smem_k]) = INT4(a[load_a_gmem_addr     + K]);
        INT4(s_b[load_b_smem_k    ][load_b_smem_n]) = INT4(b[load_b_gmem_addr        ]);
        INT4(s_b[load_b_smem_k + 1][load_b_smem_n]) = INT4(b[load_b_gmem_addr     + N]);

        load_a_gmem_addr += BK;
        load_b_gmem_addr += BK * N;

        __syncthreads();

        wmma::load_matrix_sync(frag_a[0][0], &s_a[comp_c_frag_m * 64     ][ 0], BK + APAD);
        wmma::load_matrix_sync(frag_a[0][1], &s_a[comp_c_frag_m * 64 + 16][ 0], BK + APAD);
        wmma::load_matrix_sync(frag_a[0][2], &s_a[comp_c_frag_m * 64 + 32][ 0], BK + APAD);
        wmma::load_matrix_sync(frag_a[0][3], &s_a[comp_c_frag_m * 64 + 48][ 0], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][0], &s_a[comp_c_frag_m * 64     ][16], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][1], &s_a[comp_c_frag_m * 64 + 16][16], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][2], &s_a[comp_c_frag_m * 64 + 32][16], BK + APAD);
        wmma::load_matrix_sync(frag_a[1][3], &s_a[comp_c_frag_m * 64 + 48][16], BK + APAD);

        wmma::load_matrix_sync(frag_b[0][0], &s_b[ 0][comp_c_frag_n * 64     ], BN + BPAD);
        wmma::load_matrix_sync(frag_b[0][1], &s_b[ 0][comp_c_frag_n * 64 + 16], BN + BPAD);
        wmma::load_matrix_sync(frag_b[0][2], &s_b[ 0][comp_c_frag_n * 64 + 32], BN + BPAD);
        wmma::load_matrix_sync(frag_b[0][3], &s_b[ 0][comp_c_frag_n * 64 + 48], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][0], &s_b[16][comp_c_frag_n * 64     ], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][1], &s_b[16][comp_c_frag_n * 64 + 16], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][2], &s_b[16][comp_c_frag_n * 64 + 32], BN + BPAD);
        wmma::load_matrix_sync(frag_b[1][3], &s_b[16][comp_c_frag_n * 64 + 48], BN + BPAD);

        #pragma unroll
        for (int i = 0; i < 4; i++) {
            #pragma unroll
            for (int j = 0; j < 4; j++) {
                wmma::mma_sync(frag_c[i][j], frag_a[0][i], frag_b[0][j], frag_c[i][j]);
                wmma::mma_sync(frag_c[i][j], frag_a[1][i], frag_b[1][j], frag_c[i][j]);
            }
        }

        __syncthreads();
    }

    int32_t* ch = reinterpret_cast<int32_t*>(c);
    int store_c_gmem_m = by * BM + comp_c_frag_m * 64;
    int store_c_gmem_n = bx * BN + comp_c_frag_n * 64;
    int store_c_gmem_addr = OFFSET(store_c_gmem_m, store_c_gmem_n, N);
    #pragma unroll
    for (int i = 0; i < 4; i++) {
        #pragma unroll
        for (int j = 0; j < 4; j++) {
            wmma::store_matrix_sync(&ch[store_c_gmem_addr + i * 16 * N + j * 16], frag_c[i][j], N, wmma::mem_row_major);
        }
    }
}

typedef enum{
    igemm_kernel
} IGemmTCAlgo_t;

template<IGemmTCAlgo_t algo = igemm_kernel>
void myI8I32GemmTCWarp(int8_t *a, int8_t *b, int32_t *c, int M, int N, int K) {

    if (algo == igemm_kernel) {
        const int BM = 128, BN = 128;
        dim3 blockDim(128);
        int BX = (N + BN - 1) / BN;
        int BY = (M + BM - 1) / BM;
        dim3 gridDim(BX, BY);
        igemm_basic_int8_gemm_cuda_kernel<<<gridDim, blockDim>>>(a, b, c, M, N, K);
    }
}

template <typename scalar_t1>
void printArrayByRowsToFile(const scalar_t1* arr, int M, int N, const std::string& filename) {
    std::ofstream outputFile(filename);

    if (outputFile.is_open()) {
        for (int i = 0; i < M; i++) {
            for (int j = 0; j < N; j++) {
                int index = i * N + j;
                outputFile << std::fixed << std::setprecision(6) << static_cast<int32_t>(arr[index]) << " ";
            }
            outputFile << std::endl;
        }

        outputFile.close();
    } else {
        std::cout << "Unable to open the file." << std::endl;
    }
}

float testI8I32GemmMaxError(
    void (*gpuI8I32Gemm) (int8_t *, int8_t *, int32_t *, int, int, int),
    int M, int N, int K) {

    std::random_device rd;
    std::mt19937 generator(rd());

    std::uniform_int_distribution<int8_t> distribution(-127, 127);
    
    size_t size_a = M * K * sizeof(int8_t);
    size_t size_b = K * N * sizeof(int8_t);
    size_t size_c = M * N * sizeof(int32_t);

    int8_t *h_a, *h_b, *d_a, *d_b;
    int32_t *h_c, *d_c, *h_d_c;
    h_a = (int8_t *)malloc(size_a);
    h_b = (int8_t *)malloc(size_b);
    h_c = (int32_t *)malloc(size_c);
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);
    h_d_c = (int32_t *)malloc(size_c);

    srand(time(0));
    for (int i = 0; i < M * K; i++)
        h_a[i] = distribution(generator);
    for (int i = 0; i < K * N; i++)
        h_b[i] = distribution(generator);
    
    
    cpuI8I32Gemm(h_a, h_b, h_c, M, N, K);

    hipMemcpy(d_a, h_a, size_a, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size_b, hipMemcpyHostToDevice);
    gpuI8I32Gemm(d_a, d_b, d_c, M, N, K);
    hipMemcpy(h_d_c, d_c, size_c, hipMemcpyDeviceToHost);

    float max_error = 0.0;
    for (int i = 0; i < M * N; i++) {
        float this_error = abs((float)h_d_c[i] - (float)h_c[i]);
        if (max_error != max_error || this_error != this_error) // nan
            max_error = -NAN;
        else
            max_error = max(max_error, this_error);
    }

    free(h_a); free(h_b); free(h_c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c); free(h_d_c);

    return max_error;
}

__global__ void generateInt8Data(int8_t* data, int size, unsigned int seed) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    hiprandState_t state;
    hiprand_init(seed, index, 0, &state);

    if (index < size) {
        int randValue = hiprand(&state) % 255 - 127;  // 生成-127到127之间的随机数
        data[index] = static_cast<int8_t>(randValue);
    }
}

float testI8I32GemmPerformance(
    void (*gpuI8I32Gemm) (int8_t *, int8_t *, int32_t *, int, int, int),
    int M, int N, int K, int repeat) {

    size_t size_a = M * K * sizeof(int8_t);
    size_t size_b = K * N * sizeof(int8_t);
    size_t size_c = M * N * sizeof(int32_t);
    int num_a = M * K;
    int num_b = K * N;

    int8_t *d_a, *d_b;
    int32_t *d_c;
    hipMalloc(&d_a, size_a);
    hipMalloc(&d_b, size_b);
    hipMalloc(&d_c, size_c);

    dim3 blockDim(128);
    dim3 gridDima((num_a + blockDim.x - 1) / blockDim.x);
    dim3 gridDimb((num_b + blockDim.x - 1) / blockDim.x);

    generateInt8Data<<<gridDima, blockDim>>>(d_a, num_a, 0);
    generateInt8Data<<<gridDimb, blockDim>>>(d_b, num_b, 0);

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);

    for (int i = 0; i < repeat; i++) {
        gpuI8I32Gemm(d_a, d_b, d_c, M, N, K);
    }
    hipEventRecord(end);
    hipEventSynchronize(end);

    float msec, sec;
    hipEventElapsedTime(&msec, start, end);
    sec = msec / 1000.0 / repeat;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipEventDestroy(start);
    hipEventDestroy(end);

    return sec;
}

int main(){
    const int test_num = 64;
    int M_list[test_num];
    int N_list[test_num];
    int K_list[test_num];
    for (int i = 0; i < test_num; i++) {
        M_list[i] = (i + 1) * 256;
        N_list[i] = (i + 1) * 256;
        K_list[i] = (i + 1) * 256;
    }

    const int outer_repeat = 10, inner_repeat = 1;

    {
        printf("\nalgo = HGEMMAlignedV1\n");

        {
            const int M = 128, N = 128, K = 32;
            float max_error = testI8I32GemmMaxError(
                myI8I32GemmTCWarp<igemm_kernel>, M, N, K);
            printf("Max Error when M = %d, N = %d, K = %d is %f\n", M, N, K, max_error);
        }

        {
            const int M = 512, N = 512, K = 128;
            float max_error = testI8I32GemmMaxError(
                myI8I32GemmTCWarp<igemm_kernel>, M, N, K);
            printf("Max Error when M = %d, N = %d, K = %d is %f\n", M, N, K, max_error);
        }

        for (int j = 0; j < test_num; j++) {
            int M = M_list[j], N = N_list[j], K = K_list[j];

            double max_sec = 0.0;
            double min_sec = DBL_MAX;
            double total_sec = 0.0;

            for (int k = 0; k < outer_repeat; k++) {
                double this_sec = testI8I32GemmPerformance(
                    myI8I32GemmTCWarp<igemm_kernel>, M, N, K, inner_repeat);
                max_sec = max(max_sec, this_sec);
                min_sec = min(min_sec, this_sec);
                total_sec += this_sec;
            }

            double avg_sec = total_sec / outer_repeat;
            double avg_Tflops = ((double)M) * N * K * 2 / 1024 / 1024 / 1024 / 1024 / avg_sec;

            printf("M N K = %6d %6d %6d, ", M, N, K);
            printf("Time = %12.8lf %12.8lf %12.8lf s, ", min_sec, avg_sec, max_sec);
            printf("AVG Performance = %10.4lf Tflops\n", avg_Tflops);
        }
    }

    return 0;
}